#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <limits.h>

// Include para las utilidades de computación paralela
#include "cputils.h"

/**
 * Estructura antena
 */
typedef struct {
	int y;
	int x;
} Antena;
/**
 * Macro para acceder a las posiciones del mapa
 */
#define m(y,x) mapa[ (y * cols) + x ]

__global__ void gpu_init(int *mapad, int INT_MAX, int size){
	/*Identificaciones necesarios*/
	int IDX_Thread = threadIdx.x; //Identificacion del hilo en la dimension
	int IDY_Thread = threadIdx.y; //Identificacion del hilo en la dimension y
	int IDX_block =	blockIdx.x; //Identificacion del bloque en la dimension x
	int IDY_block = blockIdx.y; //Identificacion del bloque en la dimension y
	int shapeGrid_X = gridDim.x; //Numeros del bloques en la dimension x
	int threads_per_block =	blockDim.x * blockDim.y; //Numero de hilos por bloque (1 dimension)

	/*Formula para calcular la posicion*/	//Posicion del vector dependiendo del hilo y del bloque 
	int position = threads_per_block * ((IDY_block * shapeGrid_X)+IDX_block)+((IDY_Thread*blockDim.x)+IDX_Thread);

	//inicializamos
	if(position<size)
	mapad[position] = INT_MAX;
}

/**
 * Función principal
 */
int main(int nargs, char ** vargs){


	//
	// 1. LEER DATOS DE ENTRADA
	//

	// Comprobar número de argumentos
	if(nargs < 7){
		fprintf(stderr,"Uso: %s rows cols distMax nAntenas x0 y0 [x1 y1, ...]\n",vargs[0]);
		return -1;
	}

	// Leer los argumentos de entrada
	int rows = atoi(vargs[1]);
	int cols = atoi(vargs[2]);
	int distMax = atoi(vargs[3]);
	int nAntenas = atoi(vargs[4]);

	if(nAntenas<1 || nargs != (nAntenas*2+5)){
		fprintf(stderr,"Error en la lista de antenas\n");
		return -1;
	}


	// Mensaje
	printf("Calculando el número de antenas necesarias para cubrir un mapa de"
		   " (%d x %d)\ncon una distancia máxima no superior a %d "
		   "y con %d antenas iniciales\n\n",rows,cols,distMax,nAntenas);

	// Reservar memoria para las antenas
	
	// Leer antenas
	


	//
	// 2. INICIACIÓN
	//

	// Medir el tiempo
	double tiempo = cp_Wtime();

	// Crear el mapa
	int * mapa = malloc((size_t) (rows*cols) * sizeof(int) );
	//Crear y reservar la memoria DEVICE
	int *mapad;
	hipMalloc( (void**) &mapad, sizeof(int) * (int) (rows*cols));

	// Iniciar el mapa con el valor MAX INT
	
	tam = (int) ceil((float)(rows * cols)/tam);
	dim3 bloqdimfunc1(128,1);
	dim3 griddimfunc1(tam,1);
	
	/* Enviamos la matriz al dispositivo */
	hipMemcpy(mapad, mapa, sizeof(int) * (rows*cols),hipMemcpyHostToDevice);
	
	/* Llamamos a la funcion gpu_init */
	gpu_init<<<griddimfunc1, bloqdimfunc1>>>(mapad,INT_MAX,rows*cols);
	
	/* Sincronizamos para estabilizar los datos */
	hipDeviceSynchronize();
	
	/* Recibimos la matriz de Device */
	hipMemcpy(mapa, mapad, sizeof(int) * (rows*cols),hipMemcpyDeviceToHost);
	
	//
	// 4. MOSTRAR RESULTADOS
	//

	// tiempo
	tiempo = cp_Wtime() - tiempo;	

	// Salida
	printf("Time: %f\n",tiempo);
	
	/* Comprobamos si se ha realizado bien la funcion */
	
	int error=0,z;
	for(z=0;z<rows*cols;z++){
		if(mapa[z]!=INT_MAX) error=1;
	}
	if(error) printf("Algo salio mal\n");
	else printf ("Todo correcto\n");
	
	
	/* Liberamos memoria */
	hipFree(mapad);
	
	/* Liberamos el dispositivo */
	cudaResetDevice();
	return 0;
}


