#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <limits.h>

// Include para las utilidades de computación paralela
#include "cputils.h"

/**
 * Estructura antena
 */
typedef struct {
	int y;
	int x;
} Antena;
/**
 * Macro para acceder a las posiciones del mapa
 */
#define m(y,x) mapa[ (y * cols) + x ]

#define currentGPU 0

__global__ void gpu_init(int *mapad, int max, int size){
	/*Identificaciones necesarios*/
	int IDX_Thread = threadIdx.x;	/*Identificacion del hilo en la dimension*/
	int IDY_Thread = threadIdx.y;	/*Identificacion del hilo en la dimension y*/
	int IDX_block =	blockIdx.x;	/*Identificacion del bloque en la dimension x*/
	int IDY_block = blockIdx.y;	/*Identificacion del bloque en la dimension y */
	int shapeGrid_X = gridDim.x; 	/*Numeros del bloques en la dimension */ 

	int threads_per_block =	blockDim.x * blockDim.y; /* Numero de hilos por bloque (1 dimension) */

	/*Formula para calcular la posicion*/	//Posicion del vector dependiendo del hilo y del bloque 
	int position = threads_per_block * ((IDY_block * shapeGrid_X)+IDX_block)+((IDY_Thread*blockDim.x)+IDX_Thread);

	//inicializamos
	if(position<size) mapad[position] = max;
}

void print_mapa(int * mapa, int rows, int cols, Antena * a){


	if(rows > 50 || cols > 30){
		printf("Mapa muy grande para imprimir\n");
		return;
	};


	printf("Mapa [%d,%d]\n",rows,cols);
	for(int i=0; i<rows; i++){
		for(int j=0; j<cols; j++){

			int val = m(i,j);
			printf(" %6d ",val);

		}
		printf("\n");
	}
	printf("\n");
}

/**
 * Función principal
 */
int main(int nargs, char ** vargs){


	//
	// 1. LEER DATOS DE ENTRADA
	//

	// Comprobar número de argumentos
	if(nargs < 7){
		fprintf(stderr,"Uso: %s rows cols distMax nAntenas x0 y0 [x1 y1, ...]\n",vargs[0]);
		return -1;
	}

	// Leer los argumentos de entrada
	int rows = atoi(vargs[1]);
	int cols = atoi(vargs[2]);
	int distMax = atoi(vargs[3]);
	int nAntenas = atoi(vargs[4]);

	if(nAntenas<1 || nargs != (nAntenas*2+5)){
		fprintf(stderr,"Error en la lista de antenas\n");
		return -1;
	}


	// Mensaje
	printf("Calculando el número de antenas necesarias para cubrir un mapa de"
		   " (%d x %d)\ncon una distancia máxima no superior a %d "
		   "y con %d antenas iniciales\n\n",rows,cols,distMax,nAntenas);

	// Reservar memoria para las antenas
	
	// Leer antenas
	


	//
	// 2. INICIACIÓN
	//

	// Medir el tiempo
	double tiempo = cp_Wtime();

	// Crear el mapa
	int * mapa = (int *) malloc((size_t) (rows*cols) * sizeof(int) );
	
	//Crear y reservar la memoria DEVICE
	int *mapad;
	hipMalloc( (void**) &mapad, sizeof(int) * (int) (rows*cols));

	// Iniciar el mapa con el valor MAX INT
	
	int size = rows*cols;	
	int tam = (int) ceil( ((float)(rows * cols)) /size);
	dim3 bloqdimfunc1(128,1);
	dim3 griddimfunc1(tam,1);
	/* Enviamos la matriz al dispositivo */
	hipMemcpy(mapad, mapa, sizeof(int) * (rows*cols),hipMemcpyHostToDevice);
	
	/* Llamamos a la funcion gpu_init */
	gpu_init<<<griddimfunc1, bloqdimfunc1>>>(mapad,INT_MAX,size);
	
	/* Sincronizamos para estabilizar los datos */
	hipDeviceSynchronize();
	
	/* Recibimos la matriz de Device */
	hipMemcpy(mapa, mapad, sizeof(int) * (rows*cols),hipMemcpyDeviceToHost);
	print_mapa(mapa,rows,cols,NULL);
	//
	// 4. MOSTRAR RESULTADOS
	//

	// tiempo
	tiempo = cp_Wtime() - tiempo;	

	// Salida
	printf("Time: %f\n",tiempo);
	
	/* Comprobamos si se ha realizado bien la funcion */
	
	int error=0,z;
	for(z=0;z<rows*cols;z++){
		if(mapa[z]!=INT_MAX) error=1;
	}
	if(error==1) printf("Algo salio mal\n");
	else printf ("Todo correcto\n");
	
	
	/* Liberamos memoria */
	hipFree(mapad);
	
	/* Liberamos el dispositivo */
	hipDeviceReset();
	return 0;
}

